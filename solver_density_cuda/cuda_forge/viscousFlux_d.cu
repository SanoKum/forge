#include "hip/hip_runtime.h"
#include "convectiveFlux_d.cuh"

__global__ void viscousFlux_d
( 
 // mesh structure
 geom_int nCells,
 geom_int nPlanes, geom_int nNormalPlanes, geom_int* plane_cells,  
 geom_float* vol ,  geom_float* ccx ,  geom_float* ccy, geom_float* ccz,
 geom_float* pcx ,  geom_float* pcy ,  geom_float* pcz, geom_float* fx,
 geom_float* sx  ,  geom_float* sy  ,  geom_float* sz , geom_float* ss,

 flow_float mu ,  flow_float thermCond,

 // variables
//flow_float* convx , flow_float* convy , flow_float* convz,
// flow_float* diffx , flow_float* diffy , flow_float* diffz,
 flow_float* ro   ,
 flow_float* roUx  ,
 flow_float* roUy  ,
 flow_float* roUz  ,
 flow_float* roe ,
 flow_float* Ux  ,
 flow_float* Uy  ,
 flow_float* Uz  ,
 flow_float* Ps  ,
 flow_float* Ht  ,
 flow_float* sonic,
 
 flow_float* res_ro   ,
 flow_float* res_roUx  ,
 flow_float* res_roUy  ,
 flow_float* res_roUz  ,
 flow_float* res_roe   ,

 flow_float* dUxdx  , flow_float* dUxdy , flow_float* dUxdz,
 flow_float* dUydx  , flow_float* dUydy , flow_float* dUydz,
 flow_float* dUzdx  , flow_float* dUzdy , flow_float* dUzdz,
 flow_float* dTdx   , flow_float* dTdy  , flow_float* dTdz
)
{
    geom_int ip = blockDim.x*blockIdx.x + threadIdx.x;


    if (ip < nPlanes) {

        geom_int  ic0 = plane_cells[2*ip+0];
        geom_int  ic1 = plane_cells[2*ip+1];

        geom_float f = fx[ip];
        
        geom_float sxx = sx[ip];
        geom_float syy = sy[ip];
        geom_float szz = sz[ip];
        geom_float sss = ss[ip];

        flow_float ccx_0 = ccx[ic0];
        flow_float ccy_0 = ccy[ic0];
        flow_float ccz_0 = ccz[ic0];

        flow_float ccx_1 = ccx[ic1];
        flow_float ccy_1 = ccy[ic1];
        flow_float ccz_1 = ccz[ic1];

        flow_float dcc_x = ccx_1 - ccx_0;
        flow_float dcc_y = ccy_1 - ccy_0;
        flow_float dcc_z = ccz_1 - ccz_0;
        flow_float dcc   = sqrt(dcc_x*dcc_x +dcc_y*dcc_y +dcc_z*dcc_z) ;

        flow_float Uxf = f*Ux[ic0] + (1.0-f)*Ux[ic1];
        flow_float Uyf = f*Uy[ic0] + (1.0-f)*Uy[ic1];
        flow_float Uzf = f*Uz[ic0] + (1.0-f)*Uz[ic1];

        flow_float dUxdxf = f*dUxdx[ic0] + (1.0-f)*dUxdx[ic1];
        flow_float dUxdyf = f*dUxdy[ic0] + (1.0-f)*dUxdy[ic1];
        flow_float dUxdzf = f*dUxdz[ic0] + (1.0-f)*dUxdz[ic1];

        flow_float dUydxf = f*dUydx[ic0] + (1.0-f)*dUydx[ic1];
        flow_float dUydyf = f*dUydy[ic0] + (1.0-f)*dUydy[ic1];
        flow_float dUydzf = f*dUydz[ic0] + (1.0-f)*dUydz[ic1];

        flow_float dUzdxf = f*dUzdx[ic0] + (1.0-f)*dUzdx[ic1];
        flow_float dUzdyf = f*dUzdy[ic0] + (1.0-f)*dUzdy[ic1];
        flow_float dUzdzf = f*dUzdz[ic0] + (1.0-f)*dUzdz[ic1];

        flow_float dTdxf = f*dTdx[ic0] + (1.0-f)*dTdx[ic1];
        flow_float dTdyf = f*dTdy[ic0] + (1.0-f)*dTdy[ic1];
        flow_float dTdzf = f*dTdz[ic0] + (1.0-f)*dTdz[ic1];


        flow_float alpha = sss*sss/(dcc_x*sxx +dcc_y*syy +dcc_z*szz); // over relaxed

        flow_float tau_x = mu*((Ux[ic1] -Ux[ic0])/dcc)*alpha*dcc;
        tau_x += mu*(dUxdxf*(sxx-alpha*dcc_x) +dUydxf*(syy-alpha*dcc_y) +dUzdxf*(szz-alpha*dcc_z));
        tau_x += mu*(dUxdxf*sxx + dUydxf*syy + dUzdxf*szz);
        tau_x += -mu*2.0/3.0*(dUxdxf+dUydyf+dUzdzf)*sxx;

        flow_float tau_y = mu*((Uy[ic1] -Uy[ic0])/dcc)*alpha*dcc;
        tau_y += mu*(dUxdyf*(sxx-alpha*dcc_x) +dUydyf*(syy-alpha*dcc_y) +dUzdyf*(szz-alpha*dcc_z));
        tau_y += mu*(dUxdyf*sxx + dUydyf*syy + dUzdyf*szz);
        tau_y += -mu*2.0/3.0*(dUxdxf+dUydyf+dUzdzf)*syy;

        flow_float tau_z = mu*((Uz[ic1] -Uz[ic0])/dcc)*alpha*dcc;
        tau_z += mu*(dUxdzf*(sxx-alpha*dcc_x) +dUydzf*(syy-alpha*dcc_y) +dUzdzf*(szz-alpha*dcc_z));
        tau_z += mu*(dUxdzf*sxx + dUydzf*syy + dUzdzf*szz);
        tau_z += -mu*2.0/3.0*(dUxdxf+dUydyf+dUzdzf)*szz;

        flow_float res_ro_temp   = 0.0;
        flow_float res_roUx_temp = tau_x;
        flow_float res_roUy_temp = tau_y;
        flow_float res_roUz_temp = tau_z;
        flow_float res_roe_temp  = tau_x*Uxf +tau_y*Uyf +tau_z*Uzf; 
        res_roe_temp += thermCond*(dTdxf*sxx +dTdyf*syy +dTdzf*szz);

        atomicAdd(&res_ro[ic0]  , res_ro_temp);
        atomicAdd(&res_roUx[ic0], res_roUx_temp);
        atomicAdd(&res_roUy[ic0], res_roUy_temp);
        atomicAdd(&res_roUz[ic0], res_roUz_temp);
        atomicAdd(&res_roe[ic0] , res_roe_temp);

        atomicAdd(&res_ro[ic1]  , -res_ro_temp);
        atomicAdd(&res_roUx[ic1], -res_roUx_temp);
        atomicAdd(&res_roUy[ic1], -res_roUy_temp);
        atomicAdd(&res_roUz[ic1], -res_roUz_temp);
        atomicAdd(&res_roe[ic1] , -res_roe_temp);
    }

    __syncthreads();
}


void viscousFlux_d_wrapper(solverConfig& cfg , cudaConfig& cuda_cfg , mesh& msh , variables& var , matrix& mat_ns)
{
    // ------------------------------
    // *** sum over normal planes ***
    // ------------------------------
    viscousFlux_d<<<cuda_cfg.dimGrid_plane , cuda_cfg.dimBlock>>> ( 
        // mesh structure
        msh.nCells,
        msh.nPlanes , msh.nNormalPlanes , msh.map_plane_cells_d,
        var.c_d["volume"], var.c_d["ccx"], var.c_d["ccy"], var.c_d["ccz"],
        var.p_d["pcx"]   , var.p_d["pcy"], var.p_d["pcz"], var.p_d["fx"],
        var.p_d["sx"]    , var.p_d["sy"] , var.p_d["sz"] , var.p_d["ss"],  

        cfg.visc , cfg.thermCond,

        // basic variables
        //var.c_d["convx"] , var.c_d["convy"] , var.c_d["convz"] ,
        //var.c_d["diffx"] , var.c_d["diffy"] , var.c_d["diffz"] ,
        var.c_d["ro"] ,
        var.c_d["roUx"] ,
        var.c_d["roUy"] ,
        var.c_d["roUz"] ,
        var.c_d["roe"] ,
        var.c_d["Ux"]  , 
        var.c_d["Uy"]  , 
        var.c_d["Uz"]  , 
        var.c_d["P"]  , 
        var.c_d["Ht"]  , 
        var.c_d["sonic"]  , 

        var.c_d["res_ro"] ,
        var.c_d["res_roUx"] ,
        var.c_d["res_roUy"] ,
        var.c_d["res_roUz"] ,
        var.c_d["res_roe"]  ,
       
        // gradient
        var.c_d["dUxdx"] , var.c_d["dUxdy"] , var.c_d["dUxdz"],
        var.c_d["dUydx"] , var.c_d["dUydy"] , var.c_d["dUydz"],
        var.c_d["dUzdx"] , var.c_d["dUzdy"] , var.c_d["dUzdz"],

        var.c_d["dTdx"] , var.c_d["dTdy"] , var.c_d["dTdz"]
    ) ;

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}