#include "hip/hip_runtime.h"
#include "dependentVariables_d.cuh"

__global__ void dependentVariables_d
( 
 // gas properties
 flow_float gamma , flow_float cp , 

 // mesh structure
 geom_int nCells_all , geom_int nCells,

 // variables
 flow_float* ro  ,
 flow_float* roUx  ,
 flow_float* roUy  ,
 flow_float* roUz  ,
 flow_float* roe  ,

 flow_float* P   ,
 flow_float* Ht  ,
 flow_float* sonic,
 flow_float* T   ,
 flow_float* Ux  ,
 flow_float* Uy  ,
 flow_float* Uz  
)
{
    geom_int ic = blockDim.x*blockIdx.x + threadIdx.x;

    flow_float ek;
    flow_float intE;

    flow_float ro_temp;
    flow_float T_temp;
    flow_float P_temp;

    if (ic < nCells_all) {
        ro_temp = max(ro[ic], 1e-6f);

        Ux[ic] = roUx[ic]/ro_temp;
        Uy[ic] = roUy[ic]/ro_temp;
        Uz[ic] = roUz[ic]/ro_temp;

        ek = 0.5*(Ux[ic]*Ux[ic] +Uy[ic]*Uy[ic] +Uz[ic]*Uz[ic]);
        intE =(roe[ic]/ro_temp -ek);
        T_temp = max(intE/(cp/gamma), 1e-6f);
        P_temp = max((gamma-1.0)*(roe[ic]-ro_temp*ek),1e-6f);

        T[ic] = T_temp;
        P[ic] = P_temp;

        ro[ic] = ro_temp;
        roe[ic] = P_temp/(gamma-1.0) + ro_temp*ek;

        Ht[ic] = roe[ic]/ro_temp + P_temp/ro_temp;

        sonic[ic] = sqrt(gamma*P_temp/ro_temp);

    }
}


void dependentVariables_d_wrapper(solverConfig& cfg , cudaConfig& cuda_cfg , mesh& msh , variables& var)
{
    dependentVariables_d<<<cuda_cfg.dimGrid_cell , cuda_cfg.dimBlock>>> ( 
        // gas properties
        cfg.gamma , cfg.cp , 

        // mesh structure
        msh.nCells_all , msh.nCells ,

        // basic variables
        var.c_d["ro"]  , var.c_d["roUx"], var.c_d["roUy"] , var.c_d["roUz"], var.c_d["roe"] ,
        var.c_d["P"]   , var.c_d["Ht"]  , var.c_d["sonic"], var.c_d["T"], 
        var.c_d["Ux"]  , var.c_d["Uy"]  , var.c_d["Uz"] 
    ) ;
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}