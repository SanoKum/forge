#include "cuda_forge/cudaWrapper.cuh"

#include "variables.hpp"

#include <iostream>
#include <vector>
#include <list>

namespace cudaWrapper {
    // Malloc
    void cudaMalloc_wrapper(flow_float** var_d , geom_int size)
    {
        gpuErrchk( hipMalloc(var_d , size*sizeof(flow_float)) );
    };

    void cudaMalloc_wrapper(geom_int** var_d , geom_int size)
    {
        gpuErrchk( hipMalloc(var_d , size*sizeof(geom_int)) );
    };

    void cudaMemcpy_H2D_wrapper(flow_float* vec , flow_float* var_d , geom_int numEle)
    {
        gpuErrchk( hipMemcpy(var_d, vec , (size_t)(numEle*sizeof(flow_float)), hipMemcpyHostToDevice) );
    };

    void cudaMemcpy_H2D_wrapper(geom_int* vec , geom_int* var_d , geom_int numEle)
    {
        gpuErrchk( hipMemcpy(var_d, vec , (size_t)(numEle*sizeof(geom_int)), hipMemcpyHostToDevice) );
    };


    void cudaMemcpy_D2H_wrapper(flow_float* var_d , flow_float* vec , geom_int numEle)
    {
        gpuErrchk( hipMemcpy(vec, var_d, (size_t)(numEle*sizeof(flow_float)), hipMemcpyDeviceToHost) );
    };

    void cudaMemcpy_D2H_wrapper(geom_int* var_d , geom_int* vec , geom_int numEle)
    {
        gpuErrchk( hipMemcpy(vec, var_d, (size_t)(numEle*sizeof(geom_int)), hipMemcpyDeviceToHost) );
    };


    // free
//    void cudaFree_wrapper(flow_float* var_d)
//    {
//        int is_dev_ptr = is_device_pointer((const void *) var_d);
//        if (is_dev_ptr == 1) {
//            gpuErrchk( hipFree(var_d) );
//        }
//    };
//

    void cudaFree_wrapper(int* var_d)
    {
        int is_dev_ptr = is_device_pointer((const void *) var_d);
        if (is_dev_ptr == 1) {
            gpuErrchk( hipFree(var_d) );
        }
    };

    void cudaFree_wrapper(long* var_d)
    {
        int is_dev_ptr = is_device_pointer((const void *) var_d);
        if (is_dev_ptr == 1) {
            gpuErrchk( hipFree(var_d) );
        }
    };

    void cudaFree_wrapper(float* var_d)
    {
        int is_dev_ptr = is_device_pointer((const void *) var_d);
        if (is_dev_ptr == 1) {
            gpuErrchk( hipFree(var_d) );
        }
    };

    void cudaFree_wrapper(double* var_d)
    {
        int is_dev_ptr = is_device_pointer((const void *) var_d);
        if (is_dev_ptr == 1) {
            gpuErrchk( hipFree(var_d) );
        }
    };

    int is_device_pointer(const void *ptr)
    {
        int is_device_ptr = 0;
        hipPointerAttribute_t attributes;
    
        gpuErrchk(hipPointerGetAttributes(&attributes, ptr));
    
        if(attributes.devicePointer != NULL)
        {
          is_device_ptr = 1;
        }
    
        return is_device_ptr;
    }
};
